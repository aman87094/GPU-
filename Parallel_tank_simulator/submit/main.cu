#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here
#define MAX_TANKS 1000

__device__ int calculateDistance(int x1, int y1, int x2, int y2) {
    return (x1 - x2)*(x1 - x2)+(y1 - y2)*(y1 - y2);
}

__global__ void simulateGame(int M, int N, int T, int* tankX, int* tankY, int* tankHP, int* tankScore,volatile int* d_cnt,int round) {
    int tankID = blockIdx.x * blockDim.x + threadIdx.x;
    if (tankID >T || tankHP[tankID] <= 0) return;


    int targetID = (tankID + round) % T;
    if (targetID == tankID) return; 
    
    int dx = tankX[targetID] - tankX[tankID];
    int dy = tankY[targetID] - tankY[tankID];
    
    // bool blocked = false;
    int ind=-1;
    int dis=1e9;
    for (int i = 0; i < T; ++i) {
        // if (i == tankID || i == targetID) continue;
        if(i==tankID) continue;
        int dxi = tankX[i] - tankX[tankID];
        int dyi = tankY[i] - tankY[tankID];
        // && calculateDistance(tankX[tankID], tankY[tankID], tankX[i], tankY[i]) < calculateDistance(tankX[tankID], tankY[tankID], tankX[targetID], tankY[targetID])
        if (dx * dyi == dy * dxi && ((tankX[targetID]-tankX[tankID])*(tankX[i]-tankX[tankID])>=0 &&(tankY[targetID]-tankY[tankID])*(tankY[i]-tankY[tankID])>=0)) {
            int curdst = calculateDistance(tankX[tankID], tankY[tankID], tankX[i], tankY[i]);
            if(curdist<dis && tankScore[i]>0){
                ind=i;
                dis=curdist;
            }
        }
    }
    __syncthreads();
    if (ind!=-1) {
        // tankScore[tankID]++;
        atomicAdd(&tankScore[tankID], 1);
        // tankHP[targetID]--;
        atomicAdd(&d_cnt,1);
        atomicAdd(&tankHp[ind], -1);
    }
    
}

//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int* tankHp=(int*)malloc(T * sizeof (int));

    for(int i=0;i<T;i++){
        tankHp[i]=H;
        score[i]=0;
    }

    int *d_tankX, *d_tankY, *d_tankHP, *d_tankScore;
    hipMalloc(&d_tankX,T*sizeof(int));
    hipMalloc(&d_tankY,T*sizeof(int));
    hipMalloc(&d_tankHP,T*sizeof(int));
    hipMalloc(&td_tankScore,T*sizeof(int));

    hipMemcpy(d_tankX, tankX, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tankY, tankY, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tankHP, tankHP, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tankScore, tankScore, T * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (T + blockSize - 1) / blockSize;

    int cnt=0;
    int *d_cnt;
    hipMalloc(&d_cnt,sizeof(int));
    hipMemcpy(d_cnt,cnt,sizeof(int),hipMemcpyHostToDevice);
    int round=1;

    do{
        hipMemset(d_cnt,0,sizeof(int));
        hipDeviceSynchronize();
        simulateGame<<<1, T>>>(M, N, T, d_tankX, d_tankY, d_tankHP, d_tankScore,d_cnt,round);
        hipMemcpy(cnt,d_cnt,sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        round++;

    }while(cnt>=2);

    

    hipMemcpy(tankScore, d_tankScore, T * sizeof(int), hipMemcpyDeviceToHost);

    printf("Tank scores:\n");
    for (int i = 0; i < T; ++i) {
        printf("Tank %d: %d\n", i, tankScore[i]);
    }

    hipFree(d_tankX);
    hipFree(d_tankY);
    hipFree(d_tankHP);
    hipFree(d_tankScore);


    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}